#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#if CC12
  #define REAL float
#else
  #define REAL double
#endif

static void handleError(hipError_t erro) {
    if(erro != hipSuccess) {
        printf("%s\n", hipGetErrorString(erro));
        exit(EXIT_FAILURE);
    }
}

  __global__ void mamultc(REAL *a, REAL *b, REAL *c, REAL cst, int n, int size, int *ipoint) 

{
    int i, j, ii, jj, k1, k2, k3,l_diag;
    int tid = threadIdx.x + (blockIdx.x * blockDim.x)+1;
    int stride = gridDim.x * blockDim.x;
	REAL x;

    while(tid <= size) {

        REAL sum = 0.0;
		x = cst*c[tid-1];

        i = (int)(sqrt((2*tid) + 0.25) + 0.499);
        l_diag = (i*(i + 1))/2;

        /*if(l_diag < tid) {
           j = tid - l_diag;
           i = i + 1;
           }
        else {
           j = tid - l_diag + i;
        } */
        
        i = i + 1*(l_diag < tid);
        j = tid - l_diag + i*(l_diag >= tid);

       /* j = tid - (((i-1)*i)/2);*/

        ii = ipoint[i-1];
        jj = ipoint[j-1];

        for (k1 = 1; k1 <= j; k1++) {
            sum += a[ii+k1-1]*b[jj+k1-1];
        }

        for (k2 = j+1; k2 <= i; k2++) {
            int kk = ipoint[k2-1];
            sum += a[ii+k2-1]*b[kk+j-1];
        }

        for (k3 = i+1; k3 <= n; k3++) {
            int kk = ipoint[k3-1];
            sum += a[kk+i-1]*b[kk+j-1];
        }

        c[tid-1] = sum + x;
        tid += stride;
    }
}

  __global__ void mamult2(REAL *a, REAL *b, REAL *c, REAL cst, int n)

{
    int bid,
            k;

    bid = blockIdx.x + 1;
    REAL x;

    while(bid <= n) {
        int ii = ((bid-1)*bid)/2;
        int tid = threadIdx.x + 1;

        while(tid <= bid) {
            int jj = ((tid-1)*tid)/2;
            REAL sum = 0;
            
			int l;

            for (k = 1; k <= tid; k++) {
                sum += a[ii+k-1]*b[jj+k-1];
            }

            for (k = tid+1; k <= bid; k++) {
                sum += a[ii+k-1]*b[(((k-1)*k)/2) + tid-1];
            }

            for (k = bid+1; k <= n; k++) {
                int kk = (k*(k-1))/2;
                sum += a[kk+bid-1]*b[kk+tid-1];
            }

            l = tid + ((bid-1)*bid)/2;
			
			x = cst*c[l-1];
            
			c[l-1] = sum + x;

            tid += blockDim.x;
        }

        bid += gridDim.x;
    }
}

  __global__ void mamult3(REAL *a, REAL *b, REAL *c, int n1, int n2, int n) 

{
    int bid,
            k;

    bid = blockIdx.x + n1;

    while(bid <= n2) {
        int ii = ((bid-1)*bid)/2;
        int tid = threadIdx.x + 1;

        while(tid <= bid) {
            int jj = ((tid-1)*tid)/2;
            REAL sum = 0;
            int l;

            for (k = 1; k <= tid; k++) {
                sum += a[ii+k-1]*b[jj+k-1];
            }

            for (k = tid+1; k <= bid; k++) {
                int kk = (((k-1)*k)/2);
                sum += a[ii+k-1]*b[kk + tid-1];
            }

            for (k = bid+1; k <= n; k++) {
                int kk = (k*(k - 1))/2;
                sum += a[kk + bid - 1]*b[kk + tid - 1];
            }

            l = tid + ((bid-1)*bid)/2;
            c[l-1] = sum;

            tid += blockDim.x;
        }

        bid += gridDim.x;
    }
}

  extern "C" void mamult_driver(REAL *a, REAL *b, REAL *c, int n, int nn,
        int *ipoint, REAL cst, int gridx, int blockx, float *tempo, int job) 
{

    REAL *a_dev,*b_dev,*c_dev;

    int *ipoint_dev;

    size_t size_nn,
            size_n;

    /*Time events for GPU*/
    hipEvent_t inicio,
        fim;

    int i;

    size_nn = sizeof(REAL)*nn;

    size_n = sizeof(int)*n;

    /*Starting events*/
    handleError( hipEventCreate(&inicio));
    handleError( hipEventCreate(&fim));

    /*Starting to count time*/
    handleError( hipEventRecord(inicio, 0));

    /*Allocating memory to GPU*/
    handleError( hipMalloc((void **) &a_dev, size_nn));
    handleError( hipMalloc((void **) &b_dev, size_nn));
    handleError( hipMalloc((void **) &c_dev, size_nn));
    handleError( hipMalloc((void **) &ipoint_dev, size_n));

    /*Copying values to GPU*/
    handleError( hipMemcpy( a_dev, a, size_nn, hipMemcpyHostToDevice));
    handleError( hipMemcpy( b_dev, b, size_nn, hipMemcpyHostToDevice));
    handleError( hipMemset( c_dev, 0, size_nn));
    handleError( hipMemcpy( ipoint_dev, ipoint, size_n, hipMemcpyHostToDevice));

    /*Choosing the kernel call*/
    switch (job)  {
        case 0:
            mamultc <<< gridx, blockx >>> (a_dev, b_dev, c_dev, cst, n, nn, ipoint_dev);
            break;
        case 1:
            mamult2 <<< gridx, blockx >>> (a_dev, b_dev, c_dev, cst, n);
            break;
        case 2:
            for (i = 1; i+10 <= n; i += 10) {
                printf("i=%d\n", i);
                mamult3 <<< gridx, blockx >>> (a_dev, b_dev, c_dev, i, i+9, n);
            }
            printf("i=%d, n=%d\n", i, n);
            mamult3 <<< gridx, blockx >>> (a_dev, b_dev, c_dev, i, n, n);

            break;
        default:
            printf("\n(CUDA) Undefined Job\n");
    }   

    /*Taking back the results*/
    handleError( hipMemcpy(c, c_dev, size_nn, hipMemcpyDeviceToHost));

    /*calculating time spent in CUDA*/
    handleError( hipEventRecord(fim, 0));
    handleError( hipEventSynchronize(fim));
    handleError( hipEventElapsedTime(tempo, inicio, fim));

    /*deallocating memory in GPU*/
    handleError( hipFree(a_dev));
    handleError( hipFree(b_dev));
    handleError( hipFree(c_dev));
    handleError( hipFree(ipoint_dev));

    /*destroying events in CUDA*/
    handleError( hipEventDestroy(inicio));
    handleError( hipEventDestroy(fim));
}

