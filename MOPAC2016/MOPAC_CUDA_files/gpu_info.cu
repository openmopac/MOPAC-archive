/**
 * @file gpu_info.cu
 * @author Carlos Peixoto M Junior and Julio Daniel Carvalho Maia
 * @date 11/14/2013
 */
 
/*****************************************************************************/
/*                            INCLUDES                                       */
/*****************************************************************************/
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

/**
 *
 */
extern "C" void getGPUInfo(bool *hasGpu, bool *hasDouble, int *nDevices, char *name, int *name_size, 
						   size_t *totalMem, int *clockRate, int *major, int *minor)
{
    int n;
    hipError_t error = hipGetDeviceCount(&n);
    
    if(error == hipErrorNoDevice)
    {
        *hasGpu    = false;
        //*hasDouble = false;
        *nDevices  = 0;
    }
    
    else if(error == hipErrorInsufficientDriver)
    {
        *hasGpu    = false;
        //*hasDouble = false;
        *nDevices  = 0;
    }
    else
    {
        *hasGpu    = true;
        *nDevices  = n;
        hipDeviceProp_t prop;
        

	for (int i = 0; i < n; i++){

		if(hipGetDeviceProperties(&prop, i) != hipErrorInvalidDevice)
		{
		    if(prop.major >= 2)
		        hasDouble[i] = true;
		    else
		        hasDouble[i] = false;
		}
						
		strcpy(name + (i*256), prop.name);
		name_size[i] = (int)strlen(name + (i*256));
		totalMem[i] = prop.totalGlobalMem;
		clockRate[i] = prop.clockRate;
		minor[i] = prop.minor;
		major[i] = prop.major;
    
	}
}
}

extern "C" void setDevice(int idevice, bool *stat){
    hipError_t erro;
    erro = hipSetDevice(idevice);
    if (erro == hipSuccess) *stat = true;
    else *stat = false;
}

