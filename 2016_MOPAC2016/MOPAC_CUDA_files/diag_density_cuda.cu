#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>


#if MAGMA
  #include <magma.h>
#endif

#include <omp.h>

#if CC12
  #define REAL float
#else
  #define REAL double
#endif

static bool init = false;
struct coeffs{
	double alpha, beta;
	int match;
	bool isGreater;
};

typedef struct coeffs coeff;

static void handleError(hipError_t erro) {
    if(erro != hipSuccess) {
        printf("%s\n", hipGetErrorString(erro));
        exit(EXIT_FAILURE);
    }
}


__global__ void density_cuda(REAL *c, int norbs, REAL *p, int nn, int nl1, int nl2,
           int nu1, int nu2, REAL cst, REAL frac, REAL sign)
{
    int l,
        i,
        j,
        l_diag,
        k;

    REAL sum1,sum2;

    l = threadIdx.x + (blockIdx.x * blockDim.x) + 1;
    int stride = gridDim.x * blockDim.x;
    
    while(l <= nn)
    {
        i = (int)(sqrt((2*l) + 0.25) + 0.499);
        l_diag = (i*(i + 1))/2;

        /*if(l_diag < l) {
           j = l - l_diag;
           i = i + 1;
           }
        else {
           j = l - l_diag + i;
        } */
        
        i = i + 1*(l_diag < l);
        j = l - l_diag + i*(l_diag >= l);
        
        sum1 = 0.0;
        sum2 = 0.0;

        for(k = nl2; k <= nu2; k++)
        {
            sum2 += c[(k-1)*norbs + (i-1)] * c[(k-1)*norbs + (j-1)];
        }

        for(k = nl1; k <= nu1; k++)
        {
            sum1 += c[(k-1)*norbs + (i-1)] * c[(k-1)*norbs + (j-1)];
        }

        p[l-1] = (sum2*2.0 + sum1*frac)*sign;
        p[l-1] += (cst)*(i == j);

        /*p[l-1]=(REAL) j;*/

        l += stride;
    }
}

/* DENSIT driver for GPU */

    extern "C" void density_cuda_driver(REAL *c, int norbs, REAL *p, int nn, int nl1, int nl2,
           int nu1, int nu2, REAL cst, REAL frac, REAL sign, float *ttime, int nThreads, int nBlocks)

{
    //int nBlocks;
    REAL *c_dev,
          *p_dev;

    /*Time events in the GPU*/
    hipEvent_t start,
                end;
    
    size_t size_nn,
           size_norbs;

    size_nn = sizeof(REAL)*nn;
    size_norbs = sizeof(REAL)*norbs*norbs;

    
    /*Starting events*/
    handleError(hipEventCreate(&start));
    handleError(hipEventCreate(&end));

    /*Starting to count time*/
    handleError(hipEventRecord(start, 0));

    /*Allocating memory to GPU*/
    handleError(hipMalloc((void **) &c_dev, size_norbs));
    handleError(hipMalloc((void **) &p_dev, size_nn));

    /*Copying values to GPU*/
    handleError(hipMemcpy( c_dev, c, size_norbs, hipMemcpyHostToDevice));

    density_cuda <<< nBlocks, nThreads >>> (c_dev, norbs, p_dev, nn, nl1, nl2,
           nu1, nu2, cst, frac, sign);

    /*Taking back the results*/
    handleError(hipMemcpy(p, p_dev, size_nn, hipMemcpyDeviceToHost));


    /*calculating time spent in CUDA*/
    handleError(hipEventRecord(end, 0));
    handleError(hipEventSynchronize(end));
    handleError(hipEventElapsedTime(ttime, start, end));

    /*deallocating memory in GPU*/
    handleError(hipFree(c_dev));
    handleError(hipFree(p_dev));

    /*destroying events in CUDA*/
    handleError(hipEventDestroy(start));
    handleError(hipEventDestroy(end));
}
    __global__ void computeOccs(coeffs *coeffs_occ,double *eig, double *fmo, int nocc, int n, int nvirt, double bigeps,double tiny){
    	double a,b,c,d,e,alpha, beta;
    	int stride,ij,tid;
    	int match, i;
    	int isGreater;
    	__shared__ int count;
    	if (threadIdx.x == 0){
    		count = 0;
    	}
    	__syncthreads();

    	tid = threadIdx.x+1;
    	stride = blockDim.x;
    	a = eig[blockIdx.x];
    	while(tid <= nvirt){
    		ij = (nvirt*(blockIdx.x) + (tid -1))+1;
    		b = eig[tid + nocc -1];
    		c = fmo[ij-1];
    		d = a - b;

    		e = copysign(sqrt(4.0*c*c + d*d),d);
    		alpha = sqrt(0.5*(1.0 + d/e));
    		beta = (-1.0)*copysign(sqrt(1.0 - alpha*alpha),c);


    		isGreater = (int)((abs(c) > tiny) && (abs(c/d) > bigeps));

    		match = tid;


    		if(isGreater){
				i = atomicAdd(&count, 1);
				coeffs_occ[((blockIdx.x)*nvirt + (i+1))-1].isGreater = isGreater;
				coeffs_occ[((blockIdx.x)*nvirt + (i+1))-1].match = match;
				coeffs_occ[((blockIdx.x)*nvirt + (i+1))-1].alpha = alpha;
				coeffs_occ[((blockIdx.x)*nvirt + (i+1))-1].beta  = beta;
    		}
    		__syncthreads();
    		tid += stride;
    	}
    }

    __global__ void computeVirt(coeffs *coeffs_virt,double *eig, double *fmo, int nocc, int n, int nvirt, double bigeps,double tiny){
        	double a,b,c,d,e,alpha, beta;
        	int stride,ij,tid;
        	int match, i;
        	int isGreater;
        	__shared__ int count;
        	if (threadIdx.x == 0){
        		count = 0;
        	}
        	__syncthreads();

        	tid = threadIdx.x+1;
        	stride = blockDim.x;
        	a = eig[blockIdx.x + nocc];
        	while(tid <= nocc){
        		ij = (nvirt*(tid -1) + (blockIdx.x))+1;
        		b = eig[tid -1];
        		c = fmo[ij-1];
        		d = a - b;

        		e = copysign(sqrt(4.0*c*c + d*d),d);
        		alpha = sqrt(0.5*(1.0 + d/e));
        		beta = (-1.0)*copysign(sqrt(1.0 - alpha*alpha),c);


        		isGreater = (int)((abs(c) > tiny) && (abs(c/d) > bigeps));

        		match = tid;


        		if(isGreater){
    				i = atomicAdd(&count, 1);
    				coeffs_virt[((blockIdx.x)*nocc + (i+1))-1].isGreater = isGreater;
    				coeffs_virt[((blockIdx.x)*nocc + (i+1))-1].match = match;
    				coeffs_virt[((blockIdx.x)*nocc + (i+1))-1].alpha = alpha;
    				coeffs_virt[((blockIdx.x)*nocc + (i+1))-1].beta  = beta;
        		}
        		__syncthreads();
        		tid += stride;
        	}
        }

    __global__ void rotateOccs(coeffs *coeffs_occ, double *vector, double *ca0,int nocc, int n, int nvirt){

    	int tid;
    	__shared__ coeffs coeff;
    	__shared__ double vecShared[256];
    	int count;
    	int nsweeps = (n/256) +1;
    	tid = threadIdx.x;

    	for(int i = 0; i < nsweeps; i++){
    		count = 0;
	    	vecShared[threadIdx.x] = 0;

			if (tid < n) vecShared[threadIdx.x] = vector[(blockIdx.x)*n + tid];

			while(coeffs_occ[blockIdx.x*(nvirt) + count].isGreater){


				coeff.alpha = coeffs_occ[blockIdx.x*(nvirt) + count].alpha;
				coeff.beta  = coeffs_occ[blockIdx.x*(nvirt) + count].beta;
				coeff.match = coeffs_occ[blockIdx.x*(nvirt) + count].match;

				if (tid < n) vecShared[threadIdx.x] = (coeff.alpha*vecShared[threadIdx.x]) + (coeff.beta * ca0[(coeff.match-1)*n + (tid)]);
				count++;
				__syncthreads();
			}

			if (tid < n) vector[(blockIdx.x)*n + tid] = vecShared[threadIdx.x];
			__syncthreads();
			tid += 256;
    	}
    }

    __global__ void rotateVirt(coeffs *coeffs_virt, double *vector, double *ci0,int nocc, int n, int nvirt){

       	int tid, bid;
        __shared__ coeffs coeff;
        __shared__ double vecShared[256];
        int count;
        int nsweeps = (n/256) +1;
        tid = threadIdx.x;
        bid = blockIdx.x + (nocc);

        for(int i = 0; i < nsweeps; i++){
        	count = 0;
    		__syncthreads();
            vecShared[threadIdx.x] = 0;

   			if (tid < n) vecShared[threadIdx.x] = vector[(bid)*n + tid];

   			while(coeffs_virt[blockIdx.x*(nocc) + count].isGreater){
   				coeff.alpha = coeffs_virt[blockIdx.x*(nocc) + count].alpha;
   				coeff.beta  = coeffs_virt[blockIdx.x*(nocc) + count].beta;
   				coeff.match = coeffs_virt[blockIdx.x*(nocc) + count].match;

   				if (tid < n) vecShared[threadIdx.x] = (coeff.alpha * vecShared[threadIdx.x]) - (coeff.beta * ci0[(coeff.match-1)*n + (tid)]);
   				count++;
   				__syncthreads();
    		}

    		if (tid < n) vector[(bid)*n + tid] = vecShared[threadIdx.x];
    		__syncthreads();
    		tid += 256;
        }
    }

    extern "C" void diag2GPU_Driver(double *fmo, double *eig, double *vector,double *ci0, double *ca0,
    		int nocc, int lumo, int n, double bigeps, double tiny){
    	double *eig_dev, *vector_dev, *fmo_dev, *ci0_dev, *ca0_dev;
    	coeffs *coeffs_occ,*coeffs_virt;
    	int nvirt = n-nocc;

    	size_t size_eig = sizeof(double)*n;
    	size_t size_n = sizeof(double)*n*n;
    	size_t size_ci0 = sizeof(double)*n*nocc;
    	size_t size_ca0 = sizeof(double)*n*nvirt;

    	handleError(hipMalloc((void **)&coeffs_occ,sizeof(coeffs)*nocc*(nvirt)));
    	handleError(hipMalloc((void **)&coeffs_virt,sizeof(coeffs)*(nvirt)*(nocc)));
    	handleError(hipMalloc((void **)&eig_dev, sizeof(double)*n));
    	handleError(hipMalloc((void **)&fmo_dev, sizeof(double)*n*n));
    	handleError(hipMalloc((void **)&vector_dev, sizeof(double)*n*n));
    	handleError(hipMalloc((void **)&ci0_dev, size_ci0));
    	handleError(hipMalloc((void **)&ca0_dev, size_ca0));

    	handleError(hipMemset(coeffs_occ,0,sizeof(coeffs)*nocc*nvirt));
    	handleError(hipMemset(coeffs_virt,0,sizeof(coeffs)*nocc*nvirt));

    	handleError(hipMemcpy(eig_dev, eig,size_eig,hipMemcpyHostToDevice));
    	handleError(hipMemcpy(fmo_dev,fmo,size_n,hipMemcpyHostToDevice));
    	handleError(hipMemcpy(vector_dev,vector,size_n,hipMemcpyHostToDevice));
    	handleError(hipMemcpy(ci0_dev, ci0,size_ci0,hipMemcpyHostToDevice));
    	handleError(hipMemcpy(ca0_dev, ca0,size_ca0,hipMemcpyHostToDevice));


		computeOccs<<<nocc,256>>>(coeffs_occ,eig_dev,fmo_dev,nocc,n,nvirt,bigeps,tiny);
		computeVirt<<<nvirt,256>>>(coeffs_virt,eig_dev,fmo_dev,nocc,n,nvirt,bigeps,tiny);
		hipDeviceSynchronize();

		rotateOccs<<<nocc,256>>>(coeffs_occ, vector_dev, ca0_dev,nocc, n, nvirt);
		rotateVirt<<<nvirt,256>>>(coeffs_virt, vector_dev, ci0_dev, nocc, n, nvirt);


    	handleError(hipMemcpy(vector, vector_dev, sizeof(double)*n*n,hipMemcpyDeviceToHost));


    	handleError(hipFree(vector_dev));
    	handleError(hipFree(fmo_dev));
    	handleError(hipFree(eig_dev));
    	handleError(hipFree(coeffs_occ));
    	handleError(hipFree(coeffs_virt));
    	handleError(hipFree(ci0_dev));
    	handleError(hipFree(ca0_dev));

    	return;

    }

    extern "C" void diag2GPU_Driver_2gpu(double *fmo, double *eig, double *vector,double *ci0, double *ca0,
        		int nocc, int lumo, int n, double bigeps, double tiny){

    	omp_set_num_threads(2);

#pragma omp parallel
    	{
        	double *eig_dev, *vector_dev, *fmo_dev, *ci0_dev, *ca0_dev;
        	coeffs *coeffs_occ,*coeffs_virt;
        	int nvirt = n-nocc;
        	int tid;

        	size_t size_eig = sizeof(double)*n;
        	size_t size_n = sizeof(double)*n*n;
        	size_t size_ci0 = sizeof(double)*n*nocc;
        	size_t size_ca0 = sizeof(double)*n*nvirt;

			tid = omp_get_thread_num();
			hipSetDevice(tid);

			if(tid == 0){
				handleError(hipMalloc((void **)&coeffs_occ,sizeof(coeffs)*nocc*(nvirt)));
				handleError(hipMalloc((void **)&eig_dev, sizeof(double)*n));
				handleError(hipMalloc((void **)&fmo_dev, sizeof(double)*n*n));
				handleError(hipMalloc((void **)&vector_dev, sizeof(double)*n*n));
				handleError(hipMalloc((void **)&ca0_dev, size_ca0));


				handleError(hipMemset(coeffs_occ,0,sizeof(coeffs)*nocc*nvirt));
				handleError(hipMemcpy(eig_dev, eig,size_eig,hipMemcpyHostToDevice));
				handleError(hipMemcpy(fmo_dev,fmo,size_n,hipMemcpyHostToDevice));
				handleError(hipMemcpy(vector_dev,vector,size_n,hipMemcpyHostToDevice));
				handleError(hipMemcpy(ca0_dev, ca0,size_ca0,hipMemcpyHostToDevice));
			}

			else{
				handleError(hipMalloc((void **)&coeffs_virt,sizeof(coeffs)*(nvirt)*(nocc)));
				handleError(hipMalloc((void **)&eig_dev, sizeof(double)*n));
				handleError(hipMalloc((void **)&fmo_dev, sizeof(double)*n*n));
				handleError(hipMalloc((void **)&vector_dev, sizeof(double)*n*n));
				handleError(hipMalloc((void **)&ci0_dev, size_ci0));

				handleError(hipMemset(coeffs_virt,0,sizeof(coeffs)*nocc*nvirt));
				handleError(hipMemcpy(eig_dev, eig,size_eig,hipMemcpyHostToDevice));
				handleError(hipMemcpy(fmo_dev,fmo,size_n,hipMemcpyHostToDevice));
				handleError(hipMemcpy(vector_dev,vector,size_n,hipMemcpyHostToDevice));
				handleError(hipMemcpy(ci0_dev, ci0,size_ci0,hipMemcpyHostToDevice));

			}

	#pragma omp barrier
			if(tid == 0){
				computeOccs<<<nocc,256>>>(coeffs_occ,eig_dev,fmo_dev,nocc,n,nvirt,bigeps,tiny);
				hipDeviceSynchronize();
				rotateOccs<<<nocc,256>>>(coeffs_occ, vector_dev, ca0_dev,nocc, n, nvirt);

				handleError(hipMemcpy(vector, vector_dev, sizeof(double)*n*nocc,hipMemcpyDeviceToHost));
				handleError(hipFree(vector_dev));
				handleError(hipFree(fmo_dev));
				handleError(hipFree(eig_dev));
				handleError(hipFree(coeffs_occ));
				handleError(hipFree(ca0_dev));
			}

			else{
				computeVirt<<<nvirt,256>>>(coeffs_virt,eig_dev,fmo_dev,nocc,n,nvirt,bigeps,tiny);
				hipDeviceSynchronize();
				rotateVirt<<<nvirt,256>>>(coeffs_virt, vector_dev, ci0_dev, nocc, n, nvirt);

				handleError(hipMemcpy(vector + (nocc*n) , vector_dev + (nocc*n), sizeof(double)*n*nvirt,hipMemcpyDeviceToHost));
				handleError(hipFree(vector_dev));
				handleError(hipFree(fmo_dev));
				handleError(hipFree(eig_dev));
				handleError(hipFree(coeffs_virt));
				handleError(hipFree(ci0_dev));
			}
	#pragma omp barrier
        }

   return;
}

#if MAGMA
    extern "C" void MagmaDsyevd_Driver1(int ngpus, char opt1, char opt2, int n, REAL *eigenvecs, int m,
        	REAL *eigvals, REAL *work_tmp, int lwork, int *iwork_tmp,int liwork, int *info){

        	REAL *eigenvecs_dev;
        	REAL *wa;
        	if (!init){
        		magma_init();
        		init = true;
        	}
        	if (ngpus == 1){
        		wa = (REAL *)malloc(sizeof(double)*n*n);
        		handleError(cudaMalloc((void **) &eigenvecs_dev, sizeof(double)*n*n));
        		handleError(cudaMemcpy(eigenvecs_dev, eigenvecs, sizeof(double)*n*n,cudaMemcpyHostToDevice));
        		magma_dsyevd_gpu(opt1,opt2 ,n, eigenvecs_dev,m,eigvals,wa,n,work_tmp,lwork, iwork_tmp,liwork,info);
        		free(wa);
        		cudaFree(eigenvecs_dev);
        	}
        	else {
        		magma_dsyevd_m(ngpus,opt1,opt2 ,n, eigenvecs,m,eigvals,work_tmp,lwork, iwork_tmp,liwork,info);
        	}
        	return;

        }

        extern "C" void MagmaDsyevd_Driver2(int ngpus,char opt1, char opt2, int n, REAL *eigenvecs, int m,
            REAL *eigvals, double *work, int lwork, int *iwork,int liwork, int *info){
        	REAL *eigenvecs_dev;
        	REAL *wa;

        	if (!init){
        		magma_init();
        		init = true;
        	}

        	if (ngpus == 1){
        		wa =  (REAL *)malloc(sizeof(double)*n*n);
        		handleError(cudaMalloc((void **) &eigenvecs_dev, sizeof(double)*n*n));
        		handleError(cudaMemcpy(eigenvecs_dev, eigenvecs, sizeof(double)*n*n,cudaMemcpyHostToDevice));
        		magma_dsyevd_gpu(opt1, opt2, n, eigenvecs_dev,m,eigvals,wa,n,work,lwork, iwork,liwork,info);
        		handleError(cudaMemcpy(eigenvecs,eigenvecs_dev,sizeof(double)*n*n,cudaMemcpyDeviceToHost));
        		free(wa);
        		cudaFree(eigenvecs_dev);
        	}
        	else {
        		magma_dsyevd_m(ngpus,opt1, opt2, n, eigenvecs,m,eigvals,work,lwork, iwork,liwork,info);
        	}
        	return;

        }

#endif

