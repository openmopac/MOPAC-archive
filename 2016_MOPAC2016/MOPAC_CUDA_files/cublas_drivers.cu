#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include "omp.h"

//#include "C_codes_for_GPU.cuh"


#if CC12
  #define REAL float
#else
  #define REAL double
#endif


// ============================================================================
static void handleError(hipError_t erro) {
    if(erro != hipSuccess) {
        printf("%s\n", hipGetErrorString(erro));
        exit(EXIT_FAILURE);
    }
}

// ============================================================================
// Driver for (t)asum_cublas

  extern "C" void call_asum_cublas(int n, REAL *vecx, int incx, REAL *res)

{
	hipblasHandle_t handle;
	hipblasStatus_t status;

	status = hipblasCreate(&handle);

	if(status != HIPBLAS_STATUS_SUCCESS){
		printf("Starting CUBLAS has failed  \n");
		return;
	}

	size_t size_matrix = sizeof(REAL)*n;	
	REAL *vecx_dev;

	handleError(hipMalloc((void **) &vecx_dev,size_matrix));
	
	handleError(hipMemcpy(vecx_dev,vecx,size_matrix,hipMemcpyHostToDevice));
			
#if CC12
	//printf("calling SASUM\n");		
	hipblasSasum(handle,n,vecx_dev,incx,res);
#else	
	hipblasDasum(handle,n,vecx_dev,incx,res);
#endif

	handleError(hipFree(vecx_dev));

	status = hipblasDestroy(handle);

	if (status != HIPBLAS_STATUS_SUCCESS){
		printf("Ending CUBLAS has failed \n");
		return;
	}
}

// ============================================================================
// Driver for (t)axpy_cublas 

  extern "C" void call_axpy_cublas(int n,REAL alpha, REAL *vecx, int incx, REAL *vecy,int incy)

{
	hipblasHandle_t handle;
	hipblasStatus_t status;

	status = hipblasCreate(&handle);

	if(status != HIPBLAS_STATUS_SUCCESS){
		printf("Starting CUBLAS has failed  \n");
		return;
	}

	size_t size_matrix = sizeof(REAL)*n;	
	REAL *vecx_dev,*vecy_dev;

	handleError(hipMalloc((void **) &vecx_dev,size_matrix));
	handleError(hipMalloc((void **) &vecy_dev,size_matrix));
	
	handleError(hipMemcpy(vecx_dev,vecx,size_matrix,hipMemcpyHostToDevice));
	handleError(hipMemcpy(vecy_dev,vecy,size_matrix,hipMemcpyHostToDevice));	
			
#if CC12		
	hipblasSaxpy(handle,n,&alpha,vecx_dev,incx,vecy_dev,incy);
#else		
	hipblasDaxpy(handle,n,&alpha,vecx_dev,incx,vecy_dev,incy);
#endif

	handleError(hipMemcpy(vecy,vecy_dev,size_matrix,hipMemcpyDeviceToHost));

	handleError(hipFree(vecy_dev));
	handleError(hipFree(vecx_dev));

	status = hipblasDestroy(handle);

	if (status != HIPBLAS_STATUS_SUCCESS){
		printf("Ending CUBLAS has failed \n");
		return;
	}
}

// ============================================================================
// Driver for (t)copy_cublas 

extern "C" void call_copy_cublas(int n,REAL *vecx, int incx, REAL *vecy,int incy)

{
	hipblasHandle_t handle;
	hipblasStatus_t status;

	status = hipblasCreate(&handle);

	if(status != HIPBLAS_STATUS_SUCCESS){
		printf("Starting CUBLAS has failed  \n");
		return;
	}

	size_t size_matrix = sizeof(REAL)*n;	
	REAL *vecx_dev,*vecy_dev;

	handleError(hipMalloc((void **) &vecx_dev,size_matrix));
	handleError(hipMalloc((void **) &vecy_dev,size_matrix));
	
	handleError(hipMemcpy(vecx_dev,vecx,size_matrix,hipMemcpyHostToDevice));
	handleError(hipMemcpy(vecy_dev,vecy,size_matrix,hipMemcpyHostToDevice));	
			
#if CC12		
	hipblasScopy(handle,n,vecx_dev,incx,vecy_dev,incy);	
#else		
	hipblasDcopy(handle,n,vecx_dev,incx,vecy_dev,incy);
#endif
	
	handleError(hipMemcpy(vecy,vecy_dev,size_matrix,hipMemcpyDeviceToHost));

	handleError(hipFree(vecy_dev));
	handleError(hipFree(vecx_dev));

	status = hipblasDestroy(handle);

	if (status != HIPBLAS_STATUS_SUCCESS){
		printf("Ending CUBLAS has failed \n");
		return;
	}
}

// ============================================================================
// Driver for (t)dot_cublas  

extern "C" void call_dot_cublas(int n,REAL *vecx, int incx, REAL *vecy,int incy, REAL *res)

{
	hipblasHandle_t handle;
	hipblasStatus_t status;

	status = hipblasCreate(&handle);

	if(status != HIPBLAS_STATUS_SUCCESS){
		printf("Starting CUBLAS has failed  \n");
		return;
	}

	size_t size_matrix = sizeof(REAL)*n;	
	REAL *vecx_dev,*vecy_dev;

	handleError(hipMalloc((void **) &vecx_dev,size_matrix));
	handleError(hipMalloc((void **) &vecy_dev,size_matrix));
	
	handleError(hipMemcpy(vecx_dev,vecx,size_matrix,hipMemcpyHostToDevice));
	handleError(hipMemcpy(vecy_dev,vecy,size_matrix,hipMemcpyHostToDevice));	
			
#if CC12	
	hipblasSdot(handle,n,vecx_dev,incx,vecy_dev,incy,res);
#else		
	hipblasDdot(handle,n,vecx_dev,incx,vecy_dev,incy,res);
#endif

	handleError(hipFree(vecy_dev));
	handleError(hipFree(vecx_dev));

	status = hipblasDestroy(handle);

	if (status != HIPBLAS_STATUS_SUCCESS){
		printf("Ending CUBLAS has failed \n");
		return;
	}
}

// ============================================================================
// Driver for (t)gemm_cublas 

extern "C" void call_gemm_cublas(char tra, char trb, int m,int n, int k, REAL alpha, 
	  REAL *a, int lda, REAL *b, int ldb, REAL beta, REAL *c, int ldc)

{
	hipblasHandle_t handle;
	hipblasStatus_t status;
	hipblasOperation_t transa,transb;
	status = hipblasCreate(&handle);
	REAL *a_dev,*b_dev,*c_dev;
	size_t size_matrix_a,size_matrix_b,size_matrix_c;


	if(tra == 'N' && trb == 'N'){
		transa = HIPBLAS_OP_N;
		transb = HIPBLAS_OP_N;
		size_matrix_a = sizeof(REAL)*lda*k;
		size_matrix_b = sizeof(REAL)*ldb*n;
		size_matrix_c = sizeof(REAL)*ldc*n;
	}

	else if(tra == 'N' && trb == 'T'){
		transa = HIPBLAS_OP_N;
		transb = HIPBLAS_OP_T;
		size_matrix_a = sizeof(REAL)*lda*k;
		size_matrix_b = sizeof(REAL)*ldb*k;
		size_matrix_c = sizeof(REAL)*ldc*n;
	}
	else if(tra == 'T' && trb == 'N'){
		transa = HIPBLAS_OP_T;
		transb = HIPBLAS_OP_N;
		size_matrix_a = sizeof(REAL)*lda*m;
		size_matrix_b = sizeof(REAL)*ldb*n;
		size_matrix_c = sizeof(REAL)*ldc*n;
	}

	else if(tra == 'T' && trb == 'T'){
		transa = HIPBLAS_OP_T;
		transb = HIPBLAS_OP_T;
		size_matrix_a = sizeof(REAL)*lda*m;
		size_matrix_b = sizeof(REAL)*ldb*k;
		size_matrix_c = sizeof(REAL)*ldc*n;
	}

	handleError(hipMalloc((void **) &a_dev, size_matrix_a));
	handleError(hipMalloc((void **) &b_dev, size_matrix_b));
	handleError(hipMalloc((void **) &c_dev, size_matrix_c));
	handleError(hipMemcpyAsync(a_dev,a,size_matrix_a,hipMemcpyHostToDevice));
	handleError(hipMemcpyAsync(b_dev,b,size_matrix_b,hipMemcpyHostToDevice));
	handleError(hipMemcpyAsync(c_dev,c,size_matrix_c,hipMemcpyHostToDevice));

	hipblasDgemm(handle, transa,transb,m,n,k,&alpha,a_dev,lda,b_dev,ldb,&beta,c_dev,ldc);


	handleError(hipMemcpyAsync(c,c_dev,size_matrix_c,hipMemcpyDeviceToHost));
	handleError(hipFree(a_dev));
	handleError(hipFree(b_dev));
	handleError(hipFree(c_dev));
	status = hipblasDestroy(handle);

	if(status != HIPBLAS_STATUS_SUCCESS){
		printf("Ending CUBLAS has failed \n");
			return;
	}
}

extern "C" void call_gemm_cublas_mgpu(char tra, char trb, int m,int n, int k, REAL alpha,
	  REAL *a, int lda, REAL *b, int ldb, REAL beta, REAL *c, int ldc)

{
	int ndevices;
	hipGetDeviceCount(&ndevices);
	omp_set_num_threads(ndevices);
	hipblasStatus_t status;


#pragma omp parallel shared(status)
	{
		int tid = omp_get_thread_num();
		int stride, rest, nstride, stride_out;

		hipblasHandle_t handle;
		hipblasOperation_t transa,transb;
		status = hipblasCreate(&handle);
		REAL *a_dev,*b_dev,*c_dev;
		size_t size_matrix_a;

		if(tra == 'N' && trb == 'N'){
			transa = HIPBLAS_OP_N;
			transb = HIPBLAS_OP_N;
			size_matrix_a = sizeof(REAL)*lda*k;
		}

		else if(tra == 'N' && trb == 'T'){
			transa = HIPBLAS_OP_N;
			transb = HIPBLAS_OP_T;
			size_matrix_a = sizeof(REAL)*lda*k;
		}

		else if(tra == 'T' && trb == 'N'){
			transa = HIPBLAS_OP_T;
			transb = HIPBLAS_OP_N;
			size_matrix_a = sizeof(REAL)*lda*m;
		}

		else if(tra == 'T' && trb == 'T'){
			transa = HIPBLAS_OP_T;
			transb = HIPBLAS_OP_T;
			size_matrix_a = sizeof(REAL)*lda*m;
		}


		nstride = n / ndevices;
		rest = n % ndevices;
		stride = nstride * k;
		stride_out = nstride * m;

		handleError(hipSetDevice(tid));

		handleError(hipMalloc((void **) &a_dev, size_matrix_a));
		handleError(hipMalloc((void **) &b_dev, sizeof(REAL)*stride  + (rest*k)*(tid == ndevices-1)));
		handleError(hipMalloc((void **) &c_dev, sizeof(REAL)*stride  + (rest*m)*(tid == ndevices-1)));

		handleError(hipMemcpyAsync(a_dev,a,size_matrix_a,hipMemcpyHostToDevice));
		handleError(hipMemcpyAsync(b_dev,b+(stride*tid),sizeof(REAL)*(stride + (rest*k)*(tid == ndevices-1)),hipMemcpyHostToDevice));
		handleError(hipMemcpyAsync(c_dev,c+(stride*tid),sizeof(REAL)*(stride + (rest*m)*(tid == ndevices-1)),hipMemcpyHostToDevice));

		hipblasDgemm(handle, transa,transb,m,nstride + rest*(tid == ndevices-1),k,&alpha,a_dev,lda,b_dev,ldb,&beta,c_dev,ldc);

		handleError(hipMemcpy(c + (stride_out*tid) ,c_dev,sizeof(REAL)*(stride_out + (rest*m)*(tid == ndevices-1)),hipMemcpyDeviceToHost));
		handleError(hipFree(a_dev));
		handleError(hipFree(b_dev));
		handleError(hipFree(c_dev));

		status = hipblasDestroy(handle);

	}
	if(status != HIPBLAS_STATUS_SUCCESS){
		printf("Ending CUBLAS has failed \n");
		return;
	}
}


// ============================================================================
// Driver for (t)rot_cublas

  extern "C" void call_rot_cublas(int n,REAL *veci,int k,REAL *vecj,int l,REAL alpha,REAL beta)

{
	hipblasHandle_t handle;
	hipblasStatus_t status;

	status = hipblasCreate(&handle);

	if(status != HIPBLAS_STATUS_SUCCESS){
		printf("Starting CUBLAS has failed  \n");
		return;
	}

	size_t size_vec = sizeof(REAL)*n;	
	REAL *vecj_dev,*veci_dev;

	handleError(hipMalloc((void **) &vecj_dev,size_vec));
	handleError(hipMalloc((void **) &veci_dev,size_vec));
	
	handleError(hipMemcpy(vecj_dev,vecj,size_vec,hipMemcpyHostToDevice));
	handleError(hipMemcpy(veci_dev,veci,size_vec,hipMemcpyHostToDevice));	
			
#if CC12
	hipblasSrot(handle,n,veci_dev,k,vecj_dev,l,&alpha,&beta);
#else
	hipblasDrot(handle,n,veci_dev,k,vecj_dev,l,&alpha,&beta);
#endif

	handleError(hipMemcpy(veci,veci_dev,size_vec,hipMemcpyDeviceToHost));
	handleError(hipMemcpy(vecj,vecj_dev,size_vec,hipMemcpyDeviceToHost));

	handleError(hipFree(veci_dev));
	handleError(hipFree(vecj_dev));

	status = hipblasDestroy(handle);

	if (status != HIPBLAS_STATUS_SUCCESS){
		printf("Ending CUBLAS has failed \n");
		return;
	}
}

// ============================================================================
// Driver for (t)gemv_cublas 

extern "C" void call_gemv_cublas(char tra, int m,int n, REAL alpha, REAL *a, int lda, 
             REAL *vecx, int incx, REAL beta, REAL *vecy, int incy)

{
	hipblasHandle_t handle;
	hipblasStatus_t status;
	hipblasOperation_t transa;
	status = hipblasCreate(&handle);

	if (status != HIPBLAS_STATUS_SUCCESS){
		printf("Starting CUBLAS has failed \n");
		return;
	}

	size_t size_matrix_a, size_vec_x, size_vec_y;

    REAL *a_dev,*vecx_dev,*vecy_dev;

	if(tra == 'N' ){
		transa = HIPBLAS_OP_N;

        size_matrix_a = sizeof(REAL)*lda*n;
		size_vec_x = sizeof(REAL)*n;
	    size_vec_y = sizeof(REAL)*m;
	}
	
	if(tra == 'T' ){
		transa = HIPBLAS_OP_T;
        size_matrix_a = sizeof(REAL)*lda*m;
		size_vec_x = sizeof(REAL)*m;
	    size_vec_y = sizeof(REAL)*n;
	}

	handleError(hipMalloc((void **) &a_dev,size_matrix_a));
	handleError(hipMalloc((void **) &vecx_dev,size_vec_x));
	handleError(hipMalloc((void **) &vecy_dev,size_vec_y));

	handleError(hipMemcpy(a_dev,a,size_matrix_a,hipMemcpyHostToDevice));
	handleError(hipMemcpy(vecx_dev,vecx,size_vec_x,hipMemcpyHostToDevice));
	handleError(hipMemcpy(vecy_dev,vecy,size_vec_y,hipMemcpyHostToDevice));
	
#if CC12
	hipblasSgemv(handle, transa,m,n,&alpha,a_dev,lda,vecx_dev,incx,&beta,vecy_dev,incy);
#else
	hipblasDgemv(handle, transa,m,n,&alpha,a_dev,lda,vecx_dev,incx,&beta,vecy_dev,incy);
#endif

	handleError(hipMemcpy(vecy,vecy_dev,size_vec_y,hipMemcpyDeviceToHost));
	handleError(hipFree(a_dev));
	handleError(hipFree(vecx_dev));
	handleError(hipFree(vecy_dev));

	status = hipblasDestroy(handle);
	if(status != HIPBLAS_STATUS_SUCCESS){
		printf("Ending CUBLAS has failed \n");
		return;
	}
}
  
// ============================================================================
// Driver for (t)ger_cublas 

  extern "C" void call_ger_cublas(int m,int n, REAL alpha, 
             REAL *vecx, int incx, REAL *vecy, int incy, REAL *a, int lda)

{
	hipblasHandle_t handle;
	hipblasStatus_t status;
	status = hipblasCreate(&handle);

	if (status != HIPBLAS_STATUS_SUCCESS){
		printf("Starting CUBLAS has failed \n");
		return;
	}

    size_t size_matrix_a = sizeof(REAL)*lda*n;
    size_t size_vec_x = sizeof(REAL)*m;
	size_t size_vec_y = sizeof(REAL)*n;
	REAL *a_dev,*vecx_dev,*vecy_dev;
	
	handleError(hipMalloc((void **) &a_dev,size_matrix_a));
	handleError(hipMalloc((void **) &vecx_dev,size_vec_x));
	handleError(hipMalloc((void **) &vecy_dev,size_vec_y));

	handleError(hipMemcpy(a_dev,a,size_matrix_a,hipMemcpyHostToDevice));
	handleError(hipMemcpy(vecx_dev,vecx,size_vec_x,hipMemcpyHostToDevice));
	handleError(hipMemcpy(vecy_dev,vecy,size_vec_y,hipMemcpyHostToDevice));

#if CC12
	hipblasSger(handle,m,n,&alpha,vecx_dev,incx,vecy_dev,incy,a_dev,lda);
#else
	hipblasDger(handle,m,n,&alpha,vecx_dev,incx,vecy_dev,incy,a_dev,lda);
#endif

	handleError(hipMemcpy(a,a_dev,size_matrix_a,hipMemcpyDeviceToHost));
	handleError(hipFree(a_dev));
	handleError(hipFree(vecx_dev));
	handleError(hipFree(vecy_dev));

	status = hipblasDestroy(handle);
	if(status != HIPBLAS_STATUS_SUCCESS){
		printf("Ending CUBLAS has failed \n");
		return;
	}
}

// ============================================================================
// Driver for (t)nrm2_cublas 
  
extern "C" void call_nrm2_cublas(int n,REAL *vecx, int incx, REAL *res)

{
	hipblasHandle_t handle;
	hipblasStatus_t status;

	status = hipblasCreate(&handle);

	if(status != HIPBLAS_STATUS_SUCCESS){
		printf("Starting CUBLAS has failed  \n");
		return;
	}

	size_t size_matrix = sizeof(REAL)*n;	
	REAL *vecx_dev;

	handleError(hipMalloc((void **) &vecx_dev,size_matrix));
	
	handleError(hipMemcpy(vecx_dev,vecx,size_matrix,hipMemcpyHostToDevice));
			
#if CC12	
	hipblasSnrm2(handle,n,vecx_dev,incx,res);
#else		
	hipblasDnrm2(handle,n,vecx_dev,incx,res);
#endif

	handleError(hipMemcpy(vecx,vecx_dev,size_matrix,hipMemcpyDeviceToHost));	
	
	handleError(hipFree(vecx_dev));

	status = hipblasDestroy(handle);

	if (status != HIPBLAS_STATUS_SUCCESS){
		printf("Ending CUBLAS has failed \n");
		return;
	}
}

// ============================================================================
// Driver for (t)scal_cublas 

extern "C" void call_scal_cublas(int n,REAL alpha, REAL *vecx, int incx)

{
	hipblasHandle_t handle;
	hipblasStatus_t status;

	status = hipblasCreate(&handle);

	if(status != HIPBLAS_STATUS_SUCCESS){
		printf("Starting CUBLAS has failed  \n");
		return;
	}

	size_t size_matrix = sizeof(REAL)*n;	
	REAL *vecx_dev;

	handleError(hipMalloc((void **) &vecx_dev,size_matrix));
	
	handleError(hipMemcpy(vecx_dev,vecx,size_matrix,hipMemcpyHostToDevice));
			
#if CC12		
	hipblasSscal(handle,n,&alpha, vecx_dev,incx);
#else		
	hipblasDscal(handle,n,&alpha,vecx_dev,incx);
#endif

	handleError(hipMemcpy(vecx,vecx_dev,size_matrix,hipMemcpyDeviceToHost));	
	
	handleError(hipFree(vecx_dev));

	status = hipblasDestroy(handle);

	if (status != HIPBLAS_STATUS_SUCCESS){
		printf("Ending CUBLAS has failed \n");
		return;
	}
}

// ============================================================================
// Driver for (t)swap_cublas

extern "C" void call_swap_cublas(int n,REAL *vecx, int incx, REAL *vecy,int incy)

{
	hipblasHandle_t handle;
	hipblasStatus_t status;

	status = hipblasCreate(&handle);

	if(status != HIPBLAS_STATUS_SUCCESS){
		printf("Starting CUBLAS has failed  \n");
		return;
	}

	size_t size_matrix = sizeof(REAL)*n;	
	REAL *vecx_dev,*vecy_dev;

	handleError(hipMalloc((void **) &vecx_dev,size_matrix));
	handleError(hipMalloc((void **) &vecy_dev,size_matrix));
	
	handleError(hipMemcpy(vecx_dev,vecx,size_matrix,hipMemcpyHostToDevice));
	handleError(hipMemcpy(vecy_dev,vecy,size_matrix,hipMemcpyHostToDevice));	
			
#if CC12		
	hipblasSswap(handle,n,vecx_dev,incx,vecy_dev,incy);
#else		
	hipblasDswap(handle,n,vecx_dev,incx,vecy_dev,incy);
#endif

	handleError(hipMemcpy(vecx,vecx_dev,size_matrix,hipMemcpyDeviceToHost));
	handleError(hipMemcpy(vecy,vecy_dev,size_matrix,hipMemcpyDeviceToHost));

	handleError(hipFree(vecy_dev));
	handleError(hipFree(vecx_dev));

	status = hipblasDestroy(handle);

	if (status != HIPBLAS_STATUS_SUCCESS){
		printf("Ending CUBLAS has failed \n");
		return;
	}
}

// ============================================================================
// Driver for (t)trmm_cublas 

extern "C" void call_trmm_cublas(char side, char uplo, char tra, char diag, int m, int n, 
	         REAL alpha, REAL *a, int lda, REAL *b, int ldb, REAL *c, int ldc)

{
	hipblasHandle_t handle;
	hipblasStatus_t status;
    hipblasSideMode_t side_mode;
	hipblasFillMode_t uplo_mode;
	hipblasDiagType_t diag_mode;
	hipblasOperation_t transa;
	status = hipblasCreate(&handle);

	if (status != HIPBLAS_STATUS_SUCCESS){
		printf("Starting CUBLAS has failed \n");
		return;
	}

	size_t size_matrix_a, size_matrix_b, size_matrix_c;

    REAL *a_dev,*b_dev,*c_dev;

	if (uplo == 'U' || uplo == 'u') {
		uplo_mode = HIPBLAS_FILL_MODE_UPPER;
	}

	if (uplo == 'L' || uplo == 'l') {
		uplo_mode = HIPBLAS_FILL_MODE_LOWER;
	}

    if (side == 'L' || side == 'l') {
		side_mode = HIPBLAS_SIDE_LEFT;
	    size_matrix_a = sizeof(REAL)*lda*m;
        size_matrix_b = sizeof(REAL)*ldb*n;
	    size_matrix_c = sizeof(REAL)*ldc*n;
	}

	if (side == 'R' || side == 'r') {
		side_mode = HIPBLAS_SIDE_RIGHT;
	    size_matrix_a = sizeof(REAL)*lda*n;
        size_matrix_b = sizeof(REAL)*ldb*n;
	    size_matrix_c = sizeof(REAL)*ldc*n;
	}

	if (diag == 'N' || diag == 'n') {
		diag_mode = HIPBLAS_DIAG_NON_UNIT;
	}

	if (diag == 'U' || diag == 'u') {
		diag_mode = HIPBLAS_DIAG_UNIT;
	}

	if(tra == 'N' || tra == 'n'){
		transa = HIPBLAS_OP_N;		
	}

	if(tra == 'T' || tra == 't'){
		transa = HIPBLAS_OP_T;		
	}

	//handleError( hipMalloc((void **) &c_dev, size_norbs));
	handleError( hipMalloc((void **) &a_dev, size_matrix_a));
	handleError( hipMalloc((void **) &b_dev, size_matrix_b));
    handleError( hipMalloc((void **) &c_dev, size_matrix_c));

	handleError(hipMemcpy(a_dev,a,size_matrix_a,hipMemcpyHostToDevice));
	handleError(hipMemcpy(b_dev,b,size_matrix_b,hipMemcpyHostToDevice));
    handleError(hipMemcpy(c_dev,a,size_matrix_c,hipMemcpyHostToDevice));

#if CC12
	hipblasStrmm(handle, side_mode, uplo_mode, transa,diag_mode, m,n,&alpha,a_dev,lda,
		        b_dev,ldb,c_dev,ldc);
#else
	hipblasDtrmm(handle, side_mode, uplo_mode, transa,diag_mode, m,n,&alpha,a_dev,lda,
		        b_dev,ldb,c_dev,ldc);
#endif

	handleError(hipMemcpy(c,c_dev,size_matrix_c,hipMemcpyDeviceToHost));
	handleError(hipFree(a_dev));
	handleError(hipFree(b_dev));
	handleError(hipFree(c_dev));

	status = hipblasDestroy(handle);
	if(status != HIPBLAS_STATUS_SUCCESS){
		printf("Ending CUBLAS has failed \n");
		return;
	}
}

// ============================================================================
// Driver for (t)trmv_cublas

extern "C" void call_trmv_cublas(char uplo, char tra, char diag, int n, REAL *a, int lda, 
             REAL *vecx, int incx)

{
	hipblasHandle_t handle;
	hipblasStatus_t status;
	hipblasFillMode_t uplo_mode;
	hipblasDiagType_t diag_mode;
	hipblasOperation_t transa;
	status = hipblasCreate(&handle);

	if (status != HIPBLAS_STATUS_SUCCESS){
		printf("Starting CUBLAS has failed \n");
		return;
	}

	if (uplo == 'U' || uplo == 'u') {
		uplo_mode = HIPBLAS_FILL_MODE_UPPER;
	}

	if (uplo == 'L' || uplo == 'l') {
		uplo_mode = HIPBLAS_FILL_MODE_LOWER;
	}

	if (diag == 'N' || diag == 'n') {
		diag_mode = HIPBLAS_DIAG_NON_UNIT;
	}

	if (diag == 'U' || diag == 'u') {
		diag_mode = HIPBLAS_DIAG_UNIT;
	}

	if(tra == 'N' ){
		transa = HIPBLAS_OP_N;
	}

	if(tra == 'T' ){
		transa = HIPBLAS_OP_T;
	}    

	size_t size_matrix_a = sizeof(REAL)*lda*n;
    size_t size_vec_x = sizeof(REAL)*n;
	REAL *a_dev,*vecx_dev;

	handleError(hipMalloc((void **) &a_dev,size_matrix_a));
	handleError(hipMalloc((void **) &vecx_dev,size_vec_x));

	handleError(hipMemcpy(a_dev,a,size_matrix_a,hipMemcpyHostToDevice));
	handleError(hipMemcpy(vecx_dev,vecx,size_vec_x,hipMemcpyHostToDevice));

#if CC12
	hipblasStrmv(handle, uplo_mode, transa,diag_mode,n,a_dev,lda,vecx_dev,incx);
#else
	hipblasDtrmv(handle, uplo_mode, transa,diag_mode,n,a_dev,lda,vecx_dev,incx);
#endif

	handleError(hipMemcpy(vecx,vecx_dev,size_vec_x,hipMemcpyDeviceToHost));
	handleError(hipFree(a_dev));
	handleError(hipFree(vecx_dev));

	status = hipblasDestroy(handle);
	if(status != HIPBLAS_STATUS_SUCCESS){
		printf("Ending CUBLAS has failed \n");
		return;
	}
}

// ============================================================================
// Driver for (t)trsm_cublas 

extern "C" void call_trsm_cublas(char side, char uplo, char tra, char diag, int m, 
	         int n, REAL alpha, REAL *a, int lda, REAL *b, int ldb)

{
	hipblasHandle_t handle;
	hipblasStatus_t status;
    hipblasSideMode_t side_mode;
	hipblasFillMode_t uplo_mode;
	hipblasDiagType_t diag_mode;
	hipblasOperation_t transa;
	status = hipblasCreate(&handle);

	if (status != HIPBLAS_STATUS_SUCCESS){
		printf("Starting CUBLAS has failed \n");
		return;
	}

	size_t size_matrix_a,size_matrix_b;

    REAL *a_dev,*b_dev;

	if (uplo == 'U' || uplo == 'u') {
		uplo_mode = HIPBLAS_FILL_MODE_UPPER;
	}

	if (uplo == 'L' || uplo == 'l') {
		uplo_mode = HIPBLAS_FILL_MODE_LOWER;
	}

	if (diag == 'N' || diag == 'n') {
		diag_mode = HIPBLAS_DIAG_NON_UNIT;
	}

	if (diag == 'U' || diag == 'u') {
		diag_mode = HIPBLAS_DIAG_UNIT;
	}

	if(tra == 'N' ){
		transa = HIPBLAS_OP_N;
	}

	if(tra == 'T' ){
		transa = HIPBLAS_OP_T;
	}   

    if (side == 'L' || side == 'l') {
		side_mode = HIPBLAS_SIDE_LEFT;
	    size_matrix_a = sizeof(REAL)*lda*m;
        size_matrix_b = sizeof(REAL)*ldb*n;
	}

	if (side == 'R' || side == 'r') {
		side_mode = HIPBLAS_SIDE_RIGHT;
	    size_matrix_a = sizeof(REAL)*lda*n;
        size_matrix_b = sizeof(REAL)*ldb*n;
	}

	handleError(hipMalloc((void **) &a_dev,size_matrix_a));
    handleError(hipMalloc((void **) &b_dev,size_matrix_b));

	handleError(hipMemcpy(a_dev,a,size_matrix_a,hipMemcpyHostToDevice));
	handleError(hipMemcpy(b_dev,b,size_matrix_b,hipMemcpyHostToDevice));

#if CC12
	hipblasStrsm(handle, side_mode, uplo_mode, transa,diag_mode,m,n,&alpha,a_dev,lda,b_dev,ldb);
#else
	hipblasDtrsm(handle, side_mode, uplo_mode, transa,diag_mode,m,n,&alpha,a_dev,lda,b_dev,ldb);
#endif

	handleError(hipMemcpy(b,b_dev,size_matrix_b,hipMemcpyDeviceToHost));
	handleError(hipFree(a_dev));
	handleError(hipFree(b_dev));

	status = hipblasDestroy(handle);
	if(status != HIPBLAS_STATUS_SUCCESS){
		printf("Ending CUBLAS has failed \n");
		return;
	}
}

// ============================================================================
// Driver for i(t)amax_cublas 

extern "C" void call_iamax_cublas(int n, REAL *vecx, int incx, int *res)

{
	hipblasHandle_t handle;
	hipblasStatus_t status;

	status = hipblasCreate(&handle);

	if(status != HIPBLAS_STATUS_SUCCESS){
		printf("Starting CUBLAS has failed  \n");
		return;
	}

	size_t size_matrix = sizeof(REAL)*n;	
	REAL *vecx_dev;

	handleError(hipMalloc((void **) &vecx_dev,size_matrix));
	
	handleError(hipMemcpy(vecx_dev,vecx,size_matrix,hipMemcpyHostToDevice));
			
#if CC12		
	hipblasIsamax(handle,n,vecx_dev,incx,res);
#else		
	hipblasIdamax(handle,n,vecx_dev,incx,res);
#endif
	
	handleError(hipFree(vecx_dev));

	status = hipblasDestroy(handle);

	if (status != HIPBLAS_STATUS_SUCCESS){
		printf("Ending CUBLAS has failed \n");
		return;
	}
}

  // ============================================================================
// Driver for i(t)amin_cublas 

extern "C" void call_iamin_cublas(int n, REAL *vecx, int incx, int *res)

{
	hipblasHandle_t handle;
	hipblasStatus_t status;

	status = hipblasCreate(&handle);

	if(status != HIPBLAS_STATUS_SUCCESS){
		printf("Starting CUBLAS has failed  \n");
		return;
	}

	size_t size_matrix = sizeof(REAL)*n;	
	REAL *vecx_dev;

	handleError(hipMalloc((void **) &vecx_dev,size_matrix));
	
	handleError(hipMemcpy(vecx_dev,vecx,size_matrix,hipMemcpyHostToDevice));
			
#if CC12	
	hipblasIsamin(handle,n,vecx_dev,incx,res);
#else		
	hipblasIdamin(handle,n,vecx_dev,incx,res);
#endif
	
	handleError(hipFree(vecx_dev));

	status = hipblasDestroy(handle);

	if (status != HIPBLAS_STATUS_SUCCESS){
		printf("Ending CUBLAS has failed \n");
		return;
	}
}


// ============================================================================
// Driver for (t)syrk_cublas 

extern "C" void call_syrk_cublas(char uplo, char tra, int n, int k, REAL alpha, 
			 REAL *a, int lda, REAL beta, REAL *c, int ldc)

{
	hipblasHandle_t handle;
	hipblasStatus_t status;
	hipblasFillMode_t uplo_mode;
	hipblasOperation_t transa;
	status = hipblasCreate(&handle);

	if (status != HIPBLAS_STATUS_SUCCESS){
		printf("Starting CUBLAS has failed \n");
		//return;
	}

	size_t size_matrix_a, size_matrix_c;

	REAL *a_dev,*c_dev;

	if (uplo == 'U' || uplo == 'u') {
		uplo_mode = HIPBLAS_FILL_MODE_UPPER;
	}

	if (uplo == 'L' || uplo == 'l') {
		uplo_mode = HIPBLAS_FILL_MODE_LOWER;
	}

	if(tra == 'N' || tra == 'n'){
		transa = HIPBLAS_OP_N;		
	    size_matrix_a = sizeof(REAL)*lda*k;		
	    size_matrix_c = sizeof(REAL)*ldc*n;
	}

	if(tra == 'T' || tra == 't'){
		transa = HIPBLAS_OP_T;		
	    size_matrix_a = sizeof(REAL)*lda*n;
	    size_matrix_c = sizeof(REAL)*ldc*n;
	}

	handleError(hipMalloc((void **) &a_dev,size_matrix_a));
    handleError(hipMalloc((void **) &c_dev,size_matrix_c));

	//handleError(hipMemcpy(a_dev,a,size_matrix_a,hipMemcpyHostToDevice));
        //handleError(hipMemcpy(c_dev,a,size_matrix_c,hipMemcpyHostToDevice));
	handleError(hipMemcpyAsync(a_dev,a,size_matrix_a,hipMemcpyHostToDevice));
        handleError(hipMemcpyAsync(c_dev,a,size_matrix_c,hipMemcpyHostToDevice));

#if CC12
	hipblasSsyrk(handle, uplo_mode, transa,n,k,&alpha,a_dev,lda,&beta,c_dev,ldc);
#else
	hipblasDsyrk(handle, uplo_mode, transa,n,k,&alpha,a_dev,lda,&beta,c_dev,ldc);
#endif

	//handleError(hipMemcpy(c,c_dev,size_matrix_c,hipMemcpyDeviceToHost));
	handleError(hipMemcpyAsync(c,c_dev,size_matrix_c,hipMemcpyDeviceToHost));
	handleError(hipFree(a_dev));
	handleError(hipFree(c_dev));

	status = hipblasDestroy(handle);
	if(status != HIPBLAS_STATUS_SUCCESS){
		printf("Ending CUBLAS has failed \n");
		return;
	}
}






